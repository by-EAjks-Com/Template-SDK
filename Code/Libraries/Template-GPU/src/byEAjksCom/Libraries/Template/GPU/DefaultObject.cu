#include "hip/hip_runtime.h"
//  Template-SDK, by-EAjks.Com C++ Development Best Practices
//  Copyright (c) 2022-2024 Andrea and Eric DELAGE <Contact@by-EAjks.Com>
//
//  This program is free software: you can redistribute it and/or modify
//  it under the terms of the GNU General Public License as published by
//  the Free Software Foundation, either version 3 of the License, or
//  (at your option) any later version.
//
//  This program is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.
//
//  You should have received a copy of the GNU General Public License
//  along with this program.  If not, see <https://www.gnu.org/licenses/>.

#include "byEAjksCom/Libraries/Template/GPU/DefaultObject.h"

// First-Party Headers
#include "byEAjksCom/Libraries/Template/GPU/PrecompiledHeaders.h"
#include "byEAjksCom/Libraries/Template/GPU/ObjectBase.h"

// Third-Party Headers
#include <spdlog/spdlog.h>

// Nvidia CUDA Headers
#include <hip/hip_runtime.h>

// Standard C++/STL Headers
#include <memory>

// Standard C-->C++ Headers
#include <cassert>
#include <cmath>
#include <cstdint>

__global__
void add(int n, float *x, float *y)
{
    int index  = threadIdx.x;
    int stride = blockDim.x;

    for (int i = index; i < n; i += stride)
    {
        y[i] = x[i] + y[i];
    }
}

namespace byEAjksCom::Libraries::Template::GPU {

DefaultObject::DefaultObject(
    ::std::shared_ptr<::spdlog::logger> logger)
noexcept
:   ObjectBase(::std::move(logger))
{
    this->logger->debug("entering DefaultObject::CTOR()");
}

DefaultObject::~DefaultObject()
noexcept
{
    this->logger->debug("entering DefaultObject::DTOR()");
}

auto
DefaultObject::Run()
noexcept -> void
{
    this->logger->debug("entering DefaultObject::Run()");

    ObjectBase::Run();

    int deviceCount { 0 };

    hipError_t error_id { hipGetDeviceCount(&deviceCount) };

    if (error_id != hipSuccess)
    {
        this->logger->critical(
            "hipGetDeviceCount returned {} -> {}",
            static_cast<int>(error_id),
            hipGetErrorString(error_id));

        return;
    }

    deviceCount == 0 ?
        this->logger->info("There are no available device(s) that support CUDA") :
        this->logger->info("Detected {} CUDA capable device(s)", deviceCount);

    int N { 1 << 20 };

    float * x;
    float * y;

    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int blockSize { 256 };
    int numBlocks { (N + blockSize - 1) / blockSize };
    add<<<numBlocks, blockSize>>>(N, x, y);

    hipDeviceSynchronize();

    float maxError { 0.0f };

    for (int i = 0; i < N; i++)
    {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }

    this->logger->trace("Max error: {}", maxError);

    hipFree(x);
    hipFree(y);
}

} // byEAjksCom::Libraries::Template::GPU

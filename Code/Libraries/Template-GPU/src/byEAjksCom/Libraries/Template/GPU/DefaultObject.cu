#include "hip/hip_runtime.h"
//  Template-SDK, by-EAjks.Com C++ Development Best Practices
//  Copyright (c) 2022-2024 Andrea and Eric DELAGE <Contact@by-EAjks.Com>
//
//  This program is free software: you can redistribute it and/or modify
//  it under the terms of the GNU General Public License as published by
//  the Free Software Foundation, either version 3 of the License, or
//  (at your option) any later version.
//
//  This program is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//  GNU General Public License for more details.
//
//  You should have received a copy of the GNU General Public License
//  along with this program.  If not, see <https://www.gnu.org/licenses/>.

#include "byEAjksCom/Libraries/Template/GPU/DefaultObject.h"

// Standard C-->C++ Headers
#include <cassert>
#include <cmath>
#include <cstdint>

// Standard C++/STL Headers
#include <memory>

// Nvidia CUDA Headers
#include <hip/hip_runtime.h>

// Dependency Headers
#include <spdlog/spdlog.h>

// Additional Template-SDK Headers
#include "byEAjksCom/Libraries/Template/GPU/ObjectBase.h"

__global__
void add(int n, float *x, float *y)
{
    int index  = threadIdx.x;
    int stride = blockDim.x;

    for (int i = index; i < n; i += stride)
    {
        y[i] = x[i] + y[i];
    }
}

namespace byEAjksCom::Libraries::Template::GPU {

DefaultObject::DefaultObject(
    ::std::shared_ptr<::spdlog::logger> logger)
:   ObjectBase(::std::move(logger))
{
    this->logger->debug("entering DefaultObject::CTOR()");
}

DefaultObject::~DefaultObject()
noexcept
{
    this->logger->debug("entering DefaultObject::DTOR()");
}

auto
DefaultObject::Run()
-> void
{
    this->logger->debug("entering DefaultObject::Run()");

    ObjectBase::Run();

    int deviceCount { 0 };

    hipError_t error_id { hipGetDeviceCount(&deviceCount) };

    if (error_id != hipSuccess)
    {
        this->logger->critical(
            "hipGetDeviceCount returned {} -> {}",
            static_cast<int>(error_id),
            hipGetErrorString(error_id));

        return;
    }

    deviceCount == 0 ?
        this->logger->info("There are no available device(s) that support CUDA") :
        this->logger->info("Detected {} CUDA capable device(s)", deviceCount);

    int N { 1 << 20 };

    float * x;
    float * y;

    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int blockSize { 256 };
    int numBlocks { (N + blockSize - 1) / blockSize };
    add<<<numBlocks, blockSize>>>(N, x, y);

    hipDeviceSynchronize();

    float maxError { 0.0f };

    for (int i = 0; i < N; i++)
    {
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }

    this->logger->trace("Max error: {}", maxError);

    hipFree(x);
    hipFree(y);
}

} // byEAjksCom::Libraries::Template::GPU
